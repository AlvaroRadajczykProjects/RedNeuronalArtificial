﻿#include "RedNeuronalSecuencial.cuh"

#include <stdio.h>
#include <stdlib.h>

#include <time.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

int main()
{
    srand(time(NULL));

    /*
    RedNeuronalSecuencial* r = new RedNeuronalSecuencial(4, new int[4] { 1024, 64, 64, 1024 }, new int[3] {3, 3, 3});

    float* de = new float[1024];
    float* ds = new float[1024];

    r->entrenarRedMSE_SGD(0.03, 10000, 1, 1, 1024, 1024, de, ds);
    float* res = r->propagacionHaciaDelante(1, 1024, de);
    imprimirMatrizPorPantalla("", res, 1, 1024);
    delete res;

    r->exportarRedComoArchivo("caca.data");

    delete r;

    printf("\n\ncargo el archivo:\n");

    r = new RedNeuronalSecuencial("caca.data");

    res = r->propagacionHaciaDelante(1, 1024, de);
    imprimirMatrizPorPantalla("", res, 1, 1024);
    delete res;
    delete r;
    */

    /*
    RedNeuronalSecuencial* r = new RedNeuronalSecuencial("caca.data");

    float* de = new float[1024];
    float* ds = new float[1024];

    float* res = r->propagacionHaciaDelante(1, 1024, de);
    imprimirMatrizPorPantalla("", res, 1, 1024);
    delete res;
    delete r;
    */

    const int nentradas = 2;
    const int nsalidas = 1;
    float tapren = 0.03;
    int nepochs = 10000;

    RedNeuronalSecuencial* r = new RedNeuronalSecuencial(4, new int[4] { nentradas, 10, 10, nsalidas }, new int[3] {3, 3, 3});

    const int nejemplos = 6;
    const int batch_size = 4;

    float* de = new float[nentradas * nejemplos] { 0, 0, 0, 1, 1, 0, 1, 1, 0, 0, 0, 1 };
    float* ds = new float[nsalidas * nejemplos] { 1, 0, 0, 1, 1, 0 };

    r->entrenarRedMSE_SGD(tapren, nepochs, nejemplos, batch_size, nentradas, nsalidas, de, ds);

    float* res = r->propagacionHaciaDelante(4, nentradas, de);
    imprimirMatrizPorPantalla("", res, 4, nsalidas);
    delete res;

    //r->exportarRedComoArchivo("caca.data");

    delete r;

    /*printf("\n\ncargo el archivo:\n");

    r = new RedNeuronalSecuencial("caca.data");

    res = r->propagacionHaciaDelante(4, 2, de);
    imprimirMatrizPorPantalla("", res, 4, 1);
    delete res;
    delete r;*/

    /*
    RedNeuronalSecuencial* r = new RedNeuronalSecuencial("caca.data");

    float* de = new float[8] { 0, 0, 0, 1, 1, 0, 1, 1 };
    float* ds = new float[4] { 1, 0, 0, 1 };

    float* res = r->propagacionHaciaDelante(4, 2, de);
    imprimirMatrizPorPantalla("", res, 4, 1);
    delete res;
    delete r;
    */

    return 0;
}