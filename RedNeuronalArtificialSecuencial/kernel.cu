﻿#include "RedNeuronalSecuencial.cuh"

#include <stdio.h>
#include <stdlib.h>

#include <time.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <random>

using namespace std;

int main()
{
    srand(time(NULL));

    /*
    RedNeuronalSecuencial* r = new RedNeuronalSecuencial(4, new int[4] { 1024, 64, 64, 1024 }, new int[3] {1, 1, 1});

    float* de = new float[1024];
    float* ds = new float[1024];

    //std::default_random_engine generator;
    //std::normal_distribution<float> distribution(0.0, 1.0);
    for (int i = 0; i < 1024; i++) {
        //de[i] = distribution(generator);
        //ds[i] = distribution(generator);
        de[i] =  (2 * (((float)rand()) / RAND_MAX)) - 1;
        ds[i] = (2 * (((float)rand()) / RAND_MAX)) - 1;
    }

    r->entrenarRedMSE_SGD(0.03, 10, 1000, 1, 1, 1024, 1024, de, ds);
    float* res = r->propagacionHaciaDelante(1, 1024, de);
    imprimirMatrizPorPantalla("", res, 1, 1024);
    delete res;

    r->exportarRedComoArchivo("caca.data");

    delete r;

    printf("\n\ncargo el archivo:\n");

    r = new RedNeuronalSecuencial("caca.data");

    res = r->propagacionHaciaDelante(1, 1024, de);
    imprimirMatrizPorPantalla("", res, 1, 1024);
    delete res;
    delete r;
    */

    const int nentradas = 2;
    const int nsalidas = 1;
    float tapren = 0.01;
    int nepochs = 3000;

    RedNeuronalSecuencial* r = new RedNeuronalSecuencial(5, new int[5] { nentradas, 10, 100, 10, nsalidas }, new int[4] { 1, 1, 1, 1 });

    const int nejemplos = 4;
    const int batch_size = 4;

    float* de = new float[nentradas * nejemplos] { -1, -1, -1, 1, 1, -1, 1, 1 };
    float* ds = new float[nsalidas * nejemplos] { 1, -1, -1, 1 };

    r->entrenarRedMSE_SGD(tapren, 100, nepochs, nejemplos, batch_size, nentradas, nsalidas, de, ds);

    float* res = r->propagacionHaciaDelante(4, nentradas, de);
    imprimirMatrizPorPantalla("", res, 4, nsalidas);
    delete res;
    delete r;

    /*
    RedNeuronalSecuencial* r = new RedNeuronalSecuencial("caca.data");

    float* de = new float[1024];
    float* ds = new float[1024];

    float* res = r->propagacionHaciaDelante(1, 1024, de);
    imprimirMatrizPorPantalla("", res, 1, 1024);
    delete res;
    delete r;
    */

    /*printf("\n\ncargo el archivo:\n");

    r = new RedNeuronalSecuencial("caca.data");

    res = r->propagacionHaciaDelante(4, 2, de);
    imprimirMatrizPorPantalla("", res, 4, 1);
    delete res;
    delete r;*/

    /*
    RedNeuronalSecuencial* r = new RedNeuronalSecuencial("caca.data");

    float* de = new float[8] { 0, 0, 0, 1, 1, 0, 1, 1 };
    float* ds = new float[4] { 1, 0, 0, 1 };

    float* res = r->propagacionHaciaDelante(4, 2, de);
    imprimirMatrizPorPantalla("", res, 4, 1);
    delete res;
    delete r;
    */

    return 0;
}