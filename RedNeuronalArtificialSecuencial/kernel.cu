﻿#include "RedNeuronalSecuencial.cuh"

#include <stdio.h>
#include <stdlib.h>

#include <time.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <random>

using namespace std;

int main()
{
    srand(time(NULL));

    /*
    RedNeuronalSecuencial* r = new RedNeuronalSecuencial(4, new int[4] { 1024, 64, 64, 1024 }, new int[3] {1, 1, 1});

    float* de = new float[1024];
    float* ds = new float[1024];

    //std::default_random_engine generator;
    //std::normal_distribution<float> distribution(0.0, 1.0);
    for (int i = 0; i < 1024; i++) {
        //de[i] = distribution(generator);
        //ds[i] = distribution(generator);
        de[i] =  (2 * (((float)rand()) / RAND_MAX)) - 1;
        ds[i] = (2 * (((float)rand()) / RAND_MAX)) - 1;
    }

    r->entrenarRedMSE_SGD(0.03, 10, 1000, 1, 1, 1024, 1024, de, ds);
    float* res = r->propagacionHaciaDelante(1, 1024, de);
    imprimirMatrizPorPantalla("", res, 1, 1024);
    delete res;

    r->exportarRedComoArchivo("caca.data");

    delete r;

    printf("\n\ncargo el archivo:\n");

    r = new RedNeuronalSecuencial("caca.data");

    res = r->propagacionHaciaDelante(1, 1024, de);
    imprimirMatrizPorPantalla("", res, 1, 1024);
    delete res;
    delete r;
    */

    RedNeuronalSecuencial* r; 

    const int nentradas = 2;
    const int nsalidas = 1;
    float tapren = 0.0005;
    int nepochs = 20000;
    float* res;

    const int nejemplos = 4;
    const int batch_size = 4;

    float* de = new float[nentradas * nejemplos] { 0, 0, 0, 1, 1, 0, 1, 1 };
    float* ds = new float[nsalidas * nejemplos] { 0, 1, 1, 0 };

    r = new RedNeuronalSecuencial(4, new int[4] { nentradas, 10, 10, nsalidas }, new int[3] { 3, 3, 3 });

    r->entrenarRedMSE_Adam(tapren, 0.9, 0.999, 0.000000001, 500, nepochs, nejemplos, batch_size, nentradas, nsalidas, de, ds);
    //tapren = 0.0001;
    //r->entrenarRedMSE_SGD(tapren, 100, nepochs, nejemplos, batch_size, nentradas, nsalidas, de, ds);

    r->exportarRedComoArchivo("caca.data");

    r->iniciarCublas();

    res = r->propagacionHaciaDelante(4, nentradas, de);
    imprimirMatrizPorPantalla("", res, 4, nsalidas);
    delete res;

    r->terminarCublas();

    delete r;

    r = new RedNeuronalSecuencial("caca.data");

    //r->mostrarPesosBiasesRed();

    r->iniciarCublas();

    res = r->propagacionHaciaDelante(4, nentradas, de);
    imprimirMatrizPorPantalla("", res, 4, nsalidas);
    delete res;

    r->terminarCublas();

    r->iniciarModoPropagacionDelanteRapido();

    res = new float[1] {0};

    r->propagacionDelanteRapido(new float[2] {0, 0}, res, 2);
    printf("\n%.8f", res[0]);
    r->propagacionDelanteRapido(new float[2] {0, 1}, res, 2);
    printf("\n%.8f", res[0]);
    r->propagacionDelanteRapido(new float[2] {1, 0}, res, 2);
    printf("\n%.8f", res[0]);
    r->propagacionDelanteRapido(new float[2] {1, 1}, res, 2);
    printf("\n%.8f\n", res[0]);

    r->terminarModoPropagacionDelanteRapido();

    delete r;

    /*
    RedNeuronalSecuencial* r = new RedNeuronalSecuencial("caca.data");

    float* de = new float[1024];
    float* ds = new float[1024];

    float* res = r->propagacionHaciaDelante(1, 1024, de);
    imprimirMatrizPorPantalla("", res, 1, 1024);
    delete res;
    delete r;
    */

    /*printf("\n\ncargo el archivo:\n");

    r = new RedNeuronalSecuencial("caca.data");

    res = r->propagacionHaciaDelante(4, 2, de);
    imprimirMatrizPorPantalla("", res, 4, 1);
    delete res;
    delete r;*/

    /*
    RedNeuronalSecuencial* r = new RedNeuronalSecuencial("caca.data");

    float* de = new float[8] { 0, 0, 0, 1, 1, 0, 1, 1 };
    float* ds = new float[4] { 1, 0, 0, 1 };

    float* res = r->propagacionHaciaDelante(4, 2, de);
    imprimirMatrizPorPantalla("", res, 4, 1);
    delete res;
    delete r;
    */

    return 0;
}